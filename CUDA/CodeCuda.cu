
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define initTimer struct timeval tv1, tv2; struct timezone tz
#define startTimer gettimeofday(&tv1, &tz)
#define stopTimer gettimeofday(&tv2, &tz)
#define tpsCalcul (tv2.tv_sec-tv1.tv_sec)*1000000L + (tv2.tv_usec-tv1.tv_usec)


#define MAX_DIM_GRID 65535
#define MAX_DIM_BLOCK 1024


#define MAX_CHAINE 100

#define MIN(a, b) 	(a < b ? a : b)
#define MAX(a, b) 	(a > b ? a : b)

#define CALLOC(ptr, nr, type) 		if (!(ptr = (type *) calloc((size_t)(nr), sizeof(type)))) {		\
						printf("Erreur lors de l'allocation memoire \n") ; 		\
						exit (-1);							\
					} 


#define FOPEN(fich,fichier,sens) 	if ((fich=fopen(fichier,sens)) == NULL) { 				\
						printf("Probleme d'ouverture du fichier %s\n",fichier);		\
						exit(-1);							\
					} 

#define MAX_VALEUR 	255
#define MIN_VALEUR 	0

#define NBPOINTSPARLIGNES 15

#define false 0
#define true 1
#define boolean int

long tailleVecteur ;

/* KERNEL CUDA */

__global__ void rehaussement_contraste_gpu(int *vec, int *res, int min, float coef ,long N) {
	long i = (long)blockIdx.x * (long)blockDim.x + (long)threadIdx.x;
	res[i] = (vec[i] - min) * coef;
}

void rehaussement_contraste_cpu(int *vec, int *res, int min, float coef ,long N) 
{
	long i ;
	for (i=0 ; i < N ; i ++) {
		res[i] = (vec[i] - min) * coef;
	}
}

int main(int argc, char *argv[]) {
	if (argc < 2) {
		printf("Erreur, manque un argument\n");
		exit(0);
	}
	/*========================================================================*/
	/* Declaration de variables et allocation memoire */
	/*========================================================================*/

	int i, j, n;
	
	int LE_MIN = MAX_VALEUR;
	int LE_MAX = MIN_VALEUR;
	
	float ETALEMENT = 0.0;
	
	int **image;
	int **resultat;
	int X, Y, x, y;

	int P;
	
	FILE *Src, *Dst;

	char SrcFile[MAX_CHAINE];
	char DstFile[MAX_CHAINE];
	
	char ligne[MAX_CHAINE];
	
	boolean inverse = false;
	
	char *Chemin;
	
	/*========================================================================*/
	/* Recuperation des parametres						*/
	/*========================================================================*/

	sscanf(argv[1],"%s", SrcFile);
	
	sprintf(DstFile,"%s.new",SrcFile);
	
	/*========================================================================*/
	/* Recuperation de l'endroit ou l'on travail				*/
	/*========================================================================*/

	CALLOC(Chemin, MAX_CHAINE, char);
	Chemin = getenv("PWD");
	printf("Repertoire de travail : %s \n\n",Chemin);
	

	/*========================================================================*/
	/* Ouverture des fichiers						*/
	/*========================================================================*/

	printf("Operations sur les fichiers\n");

	FOPEN(Src, SrcFile, "r");
	printf("\t Fichier source ouvert (%s) \n",SrcFile);
		
	FOPEN(Dst, DstFile, "w");
	printf("\t Fichier destination ouvert (%s) \n",DstFile);
	
	/*========================================================================*/
	/* On effectue la lecture du fichier source */
	/*========================================================================*/
	
	printf("\t Lecture entete du fichier source ");
	
	for (i = 0 ; i < 2 ; i++) {
		fgets(ligne, MAX_CHAINE, Src);	
		fprintf(Dst,"%s", ligne);
	}	

	fscanf(Src," %d %d\n",&X, &Y);
	fprintf(Dst," %d %d\n", X, Y);
	
	fgets(ligne, MAX_CHAINE, Src);	/* Lecture du 255 	*/
	fprintf(Dst,"%s", ligne);
	
	printf(": OK \n");
	
	/*========================================================================*/
	/* Allocation memoire pour l'image source et l'image resultat 		*/
	/*========================================================================*/
	
	CALLOC(image, Y+1, int *);
	CALLOC(resultat, Y+1, int *);
	for (i=0;i<Y;i++) {
		CALLOC(image[i], X+1, int);
		CALLOC(resultat[i], X+1, int);
		for (j=0;j<X;j++) {
			image[i][j] = 0;
			resultat[i][j] = 0;
		}
	}
	printf("\t\t Initialisation de l'image [%d ; %d] : Ok \n", X, Y);
			
	tailleVecteur = X * Y;
	
	x = 0;
	y = 0;
	
	
	/*========================================================================*/
	/* Lecture du fichier pour remplir l'image source 			*/
	/*========================================================================*/
	
	while (! feof(Src)) {
		n = fscanf(Src,"%d",&P);
		image[y][x] = P;	
		LE_MIN = MIN(LE_MIN, P);
		LE_MAX = MAX(LE_MAX, P);
		x ++;
		if (n == EOF || (x == X && y == Y-1)) {
			break;
		}
		if (x == X) {
			x = 0 ;
			y++;
		}
	}
	fclose(Src);
	printf("\t Lecture du fichier image : Ok \n\n");
	
	
	/*========================================================================*/
	/* Calcul du facteur d'etalement					*/
	/*========================================================================*/
	
	if (inverse) {
		ETALEMENT = 0.2;	
	} else {
		ETALEMENT = (float)(MAX_VALEUR - MIN_VALEUR) / (float)(LE_MAX - LE_MIN);	
	}
	
	/*========================================================================*/
	/* Code CUDA --> Calcul de chaque nouvelle valeur de pixel */ 
	/*========================================================================*/
	long blocksize = 1;	

	// GPUmode, if 1 -> use cuda & gpu else use cpu
	int gpumode = 1;

	int *vecteur;
	int *resultatContraste;
	int *cudaVec;
	int *cudaRes;

	initTimer;

	long size = sizeof(int)*tailleVecteur;

	vecteur = (int *)malloc(size);
	resultatContraste = (int *)malloc(size);

	if (vecteur == NULL) {
		printf("Allocation memoire qui pose probleme (vecteur) \n");
	}
	if (resultatContraste == NULL) {
		printf("Allocation memoire qui pose probleme (resultatContraste) \n");
	}

	// DONE: init vec and res
	long i_vec = 0 ;
	for (i = 0 ; i < Y ; i++) {
		for (j = 0 ; j < X ; j++) {
			vecteur[i_vec] = image[i][j];
			resultatContraste[i_vec] = 0;
			i_vec++;
		}
	}

	if (gpumode==1){
		printf("Using gpu\n");

		if (hipMalloc((void **)&cudaVec, size) == hipErrorOutOfMemory) {
			printf("Allocation memoire qui pose probleme (cudaVec) \n");
		}
		if (hipMalloc((void **)&cudaRes, size)  == hipErrorOutOfMemory) {
			printf("Allocation memoire qui pose probleme (cudaRes) \n");
		}

		long dimBlock = blocksize;
		long dimGrid = tailleVecteur/blocksize;
		if ((tailleVecteur % blocksize) != 0) {
			dimGrid++;
		}


		int res = hipMemcpy(&cudaVec[0], &vecteur[0], size, hipMemcpyHostToDevice);

		// printf("Copy CPU -> GPU %d \n",res);
	startTimer;
		rehaussement_contraste_gpu<<<dimGrid, dimBlock>>>(cudaVec, cudaRes, LE_MIN, ETALEMENT, tailleVecteur);
		// DONE: Wait for GPU to finish before accessing on host
		hipDeviceSynchronize();
	stopTimer;

		hipMemcpy(&resultatContraste[0], &cudaRes[0], size, hipMemcpyDeviceToHost);


		/* Test bon fonctionnement */

		bool ok = true;
		int indice = -1;
		int valtest = -1;
		for (i_vec= 0 ; i_vec < tailleVecteur ; i_vec++) {	
			
			valtest = (vecteur[i_vec] - LE_MIN) * ETALEMENT;
			if (resultatContraste[i_vec] != valtest) {
				// printf("Resultat GPU %d Resultat CPU %d \n", resultatContraste[i_vec], valtest);
				ok = false;
				if (indice ==-1) {
					indice = i_vec;
				}
			}
		}
		printf("------ ");
		printf("dimGrid %ld dimBlock %ld ",dimGrid, dimBlock);
		if (ok) {
			printf("Resultat ok\n");
		} else {
			printf("resultatContraste NON ok (%d)\n", indice);
		}
		
		
		hipFree(cudaVec);
		hipFree(cudaRes);
		/*========================================================================*/
		/* Fin Code CUDA */
		/*========================================================================*/

	}
	else
	{
		printf("Using cpu\n");
	startTimer;
		rehaussement_contraste_cpu(vecteur, resultatContraste, LE_MIN, ETALEMENT, tailleVecteur);
	stopTimer;

	}

	printf("chrono %ld \n", tpsCalcul);
	
	/*========================================================================*/
	/* Sauvegarde de l'image dans le fichier resultat			*/
	/*========================================================================*/
	
	n = 0;
	long cpt;
	for (cpt = 0 ; cpt < tailleVecteur ; cpt++) {
		// printf("%d \n", resultatContraste[cpt]);
		
		fprintf(Dst,"%3d ",resultatContraste[cpt]);
		n++;
		if (n == NBPOINTSPARLIGNES) {
			n = 0;
			fprintf(Dst, "\n");
		}
	}

	fprintf(Dst,"\n");
	fclose(Dst);


	/*========================================================================*/
	/* Fin du programme principal	*/
	/*========================================================================*/
	
	exit(0); 
}




