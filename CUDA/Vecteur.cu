
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define initTimer struct timeval tv1, tv2; struct timezone tz
#define startTimer gettimeofday(&tv1, &tz)
#define stopTimer gettimeofday(&tv2, &tz)
#define tpsCalcul (tv2.tv_sec-tv1.tv_sec)*1000000L + (tv2.tv_usec-tv1.tv_usec)


#define MAX_DIM_GRID 65535
#define MAX_DIM_BLOCK 1024


long tailleVecteur ;

/* KERNEL CUDA */

__global__ void add_vec_scalaire_gpu(int *vec, int *res, int a, long N) {
	long i = (long)blockIdx.x * (long)blockDim.x + (long)threadIdx.x;
	if (i < N) {
		res[i] = vec[i] + a;
	}
}

void add_vec_scalaire_cpu(int *vec, int *res, int a, long N) 
{
	int i ;
	for (i=0 ; i < N ; i ++) {
		res[i] = vec[i] + a;
	}
}

int main(int argc, char *argv[]) {
	int alpha = 10;
	if (argc < 2) {
		printf("Erreur, manque un argument\n");
		exit(0);
	}
	tailleVecteur = atol(argv[1]);
	long blocksize = 1;	
	if (argc ==3) {
		blocksize = atoi(argv[2]);
	}

	int *vecteur;
	int *resultat;
	int *cudaVec;
	int *cudaRes;

	initTimer;

	long size = sizeof(int)*tailleVecteur;

	vecteur = (int *)malloc(size);
	resultat = (int *)malloc(size);

	if (vecteur == NULL) {
		printf("Allocation memoire qui pose probleme (vecteur) \n");
	}
	if (resultat == NULL) {
		printf("Allocation memoire qui pose probleme (resultat) \n");
	}

	long i ;
	for (i= 0 ; i < tailleVecteur ; i++) {
		vecteur[i] = rand() % 100;
		resultat[i] = 0;
	}

/*	cudaSetDevice(1);	*/

	if (hipMalloc((void **)&cudaVec, size) == hipErrorOutOfMemory) {
		printf("Allocation memoire qui pose probleme (cudaVec) \n");
	}
	if (hipMalloc((void **)&cudaRes, size)  == hipErrorOutOfMemory) {
		printf("Allocation memoire qui pose probleme (cudaRes) \n");
	}

	long dimBlock = blocksize;
	long dimGrid = tailleVecteur/blocksize;
	if ((tailleVecteur % blocksize) != 0) {
		dimGrid++;
	}



	int res = hipMemcpy(&cudaVec[0], &vecteur[0], size, hipMemcpyHostToDevice);

	printf("Copy CPU -> GPU %d \n",res);
startTimer;
	add_vec_scalaire_gpu<<<dimGrid, dimBlock>>>(cudaVec, cudaRes, alpha, tailleVecteur);
	hipDeviceSynchronize();
stopTimer;

	printf("chrono_gpu %ld \n", tpsCalcul);

	hipMemcpy(&resultat[0], &cudaRes[0], size, hipMemcpyDeviceToHost);


	/* Test bon fonctionnement */

	bool ok = true;
	int indice = -1;
	for (i= 0 ; i < tailleVecteur ; i++) {
/*		printf("Resultat GPU %d     Resultat CPU %d \n",resultat[i], vecteur[i]+alpha);	*/
		if (resultat[i] != vecteur[i] + alpha) {
			ok = false;
			if (indice ==-1) {
				indice = i;
			}
		}
	}
	printf("------ ");
	printf("dimGrid %ld dimBlock %ld ",dimGrid, dimBlock);
	if (ok) {
		printf("Resultat ok\n");
	} else {
		printf("resultat NON ok (%d)\n", indice);
	}
	printf("Vecteur %ld => Temps calcul GPU %ld \n", tailleVecteur, tpsCalcul);

startTimer;
	add_vec_scalaire_cpu (vecteur, resultat, alpha, tailleVecteur);
stopTimer;
	printf("chrono_cpu %ld \n", tpsCalcul);


	hipFree(cudaVec);
	hipFree(cudaRes);


}




